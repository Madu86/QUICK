/*
  !---------------------------------------------------------------------!
  ! Created by Madu Manathunga on 04/29/2020                            !
  !                                                                     !                           
  ! Previous contributors: Yipu Miao                                    !
  !                                                                     !
  ! Copyright (C) 2020-2021 Merz lab                                    !
  ! Copyright (C) 2020-2021 Götz lab                                    !
  !                                                                     !
  ! This Source Code Form is subject to the terms of the Mozilla Public !
  ! License, v. 2.0. If a copy of the MPL was not distributed with this !
  ! file, You can obtain one at http://mozilla.org/MPL/2.0/.            !
  !_____________________________________________________________________!

  !---------------------------------------------------------------------!
  ! This source file contains device kernels pertaining to single       !
  ! precision ERI computation.                                          !
  !---------------------------------------------------------------------!
*/

#include "gpu.h"
#include <hip/hip_runtime.h>

#define SINGLE_PRECISION
#undef QUICKDouble
#define QUICKDouble float

static __constant__ int devTrans[TRANSDIM*TRANSDIM*TRANSDIM];
static __constant__ int Sumindex[10]={0,0,1,4,10,20,35,56,84,120};

#include "gpu_get2e_sp.h"
#include "gpu_get2e_subs_hrr_sp.h"
#include "int.h"


#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"
/*#include "gpu_get2e_subs_grad_sp.h"


//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"
*/

#ifdef CUDA_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"



#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"



#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"


#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "gpu_get2e_subs_sp.h"
#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

//Include the kernels for open shell eri calculations
#define OSHELL
#define int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#undef new_quick_2_gpu_get2e_subs_h
#include "gpu_get2e_subs_sp.h"
/*#include "gpu_get2e_subs_grad_sp.h"

//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_grad_sp.h"
*/

#ifdef CUDA_SPDF
//===================================

#undef int_spd
#define int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#define int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#define int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#define int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#define int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#define int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#define int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#define int_spdf8
#undef int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#define int_spdf9
#undef int_spdf10
#include "gpu_get2e_subs_sp.h"

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#define int_spdf10
#include "gpu_get2e_subs_sp.h"
#endif

#undef int_spd
#undef int_spdf
#undef int_spdf2
#undef int_spdf3
#undef int_spdf4
#undef int_spdf5
#undef int_spdf6
#undef int_spdf7
#undef int_spdf8
#undef int_spdf9
#undef int_spdf10

#undef OSHELL
#undef SINGLE_PRECISION
#undef QUICKDouble
#define QUICKDouble double
